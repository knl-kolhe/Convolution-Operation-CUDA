#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

using namespace std;

__global__ void Convolution1(int *a,int *filter,int *result,int size_a,int size_filter,int size_result)

{

        int i=blockIdx.x;
	int j=blockIdx.y;

        if(i<size_result||j<size_result)
        {
            for(int k=0;k<size_filter;k++)
                for(int l=0;l<size_filter;l++)
			result[i*size_result+j] += filter[k*size_filter+l]*a[(2*i+k)*size_a+2*j+l];
	}

}



void Convolution2(int *a,int *filter,int *result,int size_a,int size_filter,int size_result)
{
        for(int i=0;i<size_result;i++)
        {
                for(int j=0;j<size_result;j++)
					for(int k=0;k<size_filter;k++)
						for(int l=0;l<size_filter;l++)
                                result[i*size_result+j] += filter[k*size_filter+l]*a[(2*i+k)*size_a+2*j+l];
        }
}




int main()
{

        int *a,*filter,*result,*result_serial;

        int size_a,size_filter,size_result;

        clock_t t;

        double time_taken;



        x: printf("\n Enter size of array:");

        scanf("%d",&size_a);

        printf("\n Enter size of filter:");

        scanf("%d",&size_filter);

        if(size_a%2==0||size_filter%2==0)

        {

                printf("\n Enter odd numbers for sizes.");

                goto x;

        }

        if((size_a-size_filter)<0)

        {
                printf("\n Enter larger matrix size or smaller filter size.");

                goto x;

        }

        size_result=(size_a-size_filter)/2 +1;

        printf("Size of Matrix after Convolution with stride = (2) will be: %d \n",size_result);



        hipMallocManaged(&a,size_a*size_a*sizeof(int));

        hipMallocManaged(&filter,size_filter*size_filter*sizeof(int));

        hipMallocManaged(&result,size_result*size_result*sizeof(int));

        hipMallocManaged(&result_serial,size_result*size_result*sizeof(int));

        srand(0);



        for(int i=0;i<size_a*size_a;i++)
        {
                a[i]=rand()%100;
		//printf("Enter a[%d]",i);
		//scanf("%d",&a[i]);
        }

        for(int i=0;i<size_filter*size_filter;i++)
        {
                filter[i]=rand()%100;
		//printf("Enter filter[%d]",i);
		//scanf("%d",&filter[i]);
        }
        for(int i=0;i<size_result*size_result;i++)

        {

                result[i]=0;
                result_serial[i]=0;

        }
        
        //Define a block of size_result by size_result
	dim3 res(size_result,size_result);
        t=clock();
        //Call the parallel function
        Convolution1<<<res,1>>>(a,filter,result,size_a,size_filter,size_result);

        hipDeviceSynchronize();

        t=clock()-t;

        time_taken=((double)t)/CLOCKS_PER_SEC;

        printf("Time for Convolution with %d threads: %f \n",size_result*size_result,time_taken);





        t=clock();
        //Code to perform Convolution using a serial algorithm
        Convolution2(a,filter,result_serial,size_a,size_filter,size_result);

        t=clock()-t;

        time_taken=((double)t)/CLOCKS_PER_SEC;

        printf("Time for Convolution using serial:%f \n",time_taken);
		
	/* Code to print Result
	for(int i=0;i<size_result*size_result;i++)
	{
		printf("\nresult[%d]=%d \nresult_serial[%d]=%d",i,result[i],i,result_serial[i]);
	}
	*/

        hipFree(a);

        hipFree(filter);

        hipFree(result);

        hipFree(result_serial);

        return 0;

}




/***********************OUTPUT*************************

 On GTX 1050, i7 7700 4 core
 Enter size of array:10001

 Enter size of filter:3
 Size of Matrix after Convolution with stride = (2) will be: 5000
 Time for Convolution with 25000000 threads: 2.023000
 Time for Convolution using serial:3.144000

******************************************************/
